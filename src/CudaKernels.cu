#include "hip/hip_runtime.h"
#include "CudaKernels.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define DIM 3

#define BLOCK_HEIGHT 128

#define DOT_FLOPS 5
#define CROSS_FLOPS 9
#define AXPY_FLOPS 6
#define XDY_FLOPS 3

void CE() {
  hipError_t ce = hipGetLastError();
  printf("%s\n", hipGetErrorString(ce));
}

__global__
void dotProdKernel(const float* a, const float* b, int stride, int num_surfs, float* aDb) {
  unsigned int blockOff, resOff, off, res;
  float aXReg, aYReg, aZReg, bXReg, bYReg, bZReg, dotProd;
  resOff = blockIdx.x * stride;
  blockOff = resOff * DIM;

  int numChunkLoop = stride / BLOCK_HEIGHT;

  off = blockOff + threadIdx.x;
  res = resOff + threadIdx.x;
  
  for (int chunk = 0; chunk < numChunkLoop; chunk++) {
    aXReg = a[off];
    aYReg = a[off + stride];
    aZReg = a[off + stride + stride];
    bXReg = b[off];
    bYReg = b[off + stride];
    bZReg = b[off + stride + stride];
    dotProd = aXReg * bXReg + aYReg * bYReg + aZReg * bZReg;
    aDb[res] = dotProd;
    off += BLOCK_HEIGHT;
    res += BLOCK_HEIGHT;
  }

  if (off < blockOff + stride) {
    aXReg = a[off];
    aYReg = a[off + stride];
    aZReg = a[off + stride + stride];
    bXReg = b[off];
    bYReg = b[off + stride];
    bZReg = b[off + stride + stride];
    dotProd = aXReg * bXReg + aYReg * bYReg + aZReg * bZReg;
    aDb[res] = dotProd;
  }
}

void DotProductGpu(const float *a, const float *b, int stride, int num_surfs, float *aDb)
{
#ifdef GPU_PROF
    float kernelTime, flops, gflopRate;
    flops = DOT_FLOPS * stride * num_surfs;
    hipEvent_t kernelStart, kernelStop;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);
    hipEventRecord(kernelStart, 0);
#endif

    int GridDim = num_surfs;

  dotProdKernel<<<GridDim, BLOCK_HEIGHT>>>
      (a, b, stride, num_surfs, aDb);

  hipDeviceSynchronize();
#ifdef GPU_PROF
  hipEventRecord(kernelStop, 0);
  hipEventSynchronize(kernelStop);
  hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
  gflopRate = (flops / 1e9) / (kernelTime / 1e3);
  fprintf(stderr, "Dot product kernel took %f ms @ %f Gflops\n", kernelTime, gflopRate);
#endif
}

__global__
void crossProdKernel(const float* a, const float* b, int stride, int num_surfs, float* aCb) {
  unsigned int blockOff, off;
  float aXReg, aYReg, aZReg, bXReg, bYReg, bZReg, aCbXReg, aCbYReg, aCbZReg;
  blockOff = blockIdx.x * stride * DIM;

  int numChunkLoop = stride / BLOCK_HEIGHT;

  off = blockOff + threadIdx.x;
  
  for (int chunk = 0; chunk < numChunkLoop; chunk++) {
    aXReg = a[off];
    aYReg = a[off + stride];
    aZReg = a[off + stride + stride];
    bXReg = b[off];
    bYReg = b[off + stride];
    bZReg = b[off + stride + stride];

    aCbXReg = aYReg * bZReg - aZReg * bYReg;
    aCbYReg = aZReg * bXReg - aXReg * bZReg;
    aCbZReg = aXReg * bYReg - aYReg * bXReg;

    aCb[off] = aCbXReg;
    aCb[off + stride] = aCbYReg;
    aCb[off + stride + stride] = aCbZReg;

    off += BLOCK_HEIGHT;
  }

  if (off < blockOff + stride) {
    aXReg = a[off];
    aYReg = a[off + stride];
    aZReg = a[off + stride + stride];
    bXReg = b[off];
    bYReg = b[off + stride];
    bZReg = b[off + stride + stride];

    aCbXReg = aYReg * bZReg - aZReg * bYReg;
    aCbYReg = aZReg * bXReg - aXReg * bZReg;
    aCbZReg = aXReg * bYReg - aYReg * bXReg;

    aCb[off] = aCbXReg;
    aCb[off + stride] = aCbYReg;
    aCb[off + stride + stride] = aCbZReg;
  }
}

void CrossProductGpu(const float *a, const float *b, int stride, int num_surfs, float *aCb) 
{
#ifdef GPU_PROF
  float kernelTime, flops, gflopRate;
  flops = CROSS_FLOPS * stride * num_surfs;
  hipEvent_t kernelStart, kernelStop;
  hipEventCreate(&kernelStart);
  hipEventCreate(&kernelStop);
  hipEventRecord(kernelStart, 0);
#endif

  int GridDim = num_surfs;
  crossProdKernel<<<GridDim, BLOCK_HEIGHT>>>
      (a, b, stride, num_surfs, aCb);

  hipDeviceSynchronize();
#ifdef GPU_PROF
  hipEventRecord(kernelStop, 0);
  hipEventSynchronize(kernelStop);
  hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
  gflopRate = (flops / 1e9) / (kernelTime / 1e3);
  fprintf(stderr, "Cross product kernel took %f ms @ %f Gflops\n", kernelTime, gflopRate);
#endif
}

__global__
void xvpwKernel(const float *x, const float *a, const float *y, int stride, int num_surfs, float *AxPy) {
  unsigned int blockOff, scalOff, off, scal;
  float xXReg, xYReg, xZReg, aReg, yXReg, yYReg, yZReg, AxPyXReg, AxPyYReg, AxPyZReg;
  scalOff = blockIdx.x * stride;
  blockOff = scalOff * DIM;

  int numChunkLoop = stride / BLOCK_HEIGHT;

  off = blockOff + threadIdx.x;
  scal = scalOff + threadIdx.x;
  
  for (int chunk = 0; chunk < numChunkLoop; chunk++) {
    xXReg = x[off];
    xYReg = x[off + stride];
    xZReg = x[off + stride + stride];
    yXReg = y[off];
    yYReg = y[off + stride];
    yZReg = y[off + stride + stride];
    aReg = a[scal];
    

    AxPyXReg = aReg * xXReg + yXReg;
    AxPyYReg = aReg * xYReg + yYReg;
    AxPyZReg = aReg * xZReg + yZReg;

    AxPy[off] = AxPyXReg;
    AxPy[off + stride] = AxPyYReg;
    AxPy[off + stride + stride] = AxPyZReg;

    off += BLOCK_HEIGHT;
    scal += BLOCK_HEIGHT;
  }

  if (off < blockOff + stride) {
    xXReg = x[off];
    xYReg = x[off + stride];
    xZReg = x[off + stride + stride];
    yXReg = y[off];
    yYReg = y[off + stride];
    yZReg = y[off + stride + stride];
    aReg = a[scal];
    

    AxPyXReg = aReg * xXReg + yXReg;
    AxPyYReg = aReg * xYReg + yYReg;
    AxPyZReg = aReg * xZReg + yZReg;

    AxPy[off] = AxPyXReg;
    AxPy[off + stride] = AxPyYReg;
    AxPy[off + stride + stride] = AxPyZReg;
  }

}

void xvpwGpu(const float *x, const float *a, const float *y, int stride, int num_surfs, float *AxPy) {

#ifdef GPU_PROF
  float kernelTime, flops, gflopRate;
  flops = AXPY_FLOPS * stride * num_surfs;
  hipEvent_t kernelStart, kernelStop;
  hipEventCreate(&kernelStart);
  hipEventCreate(&kernelStop);
  hipEventRecord(kernelStart, 0);
#endif

  int GridDim = num_surfs;
  xvpwKernel<<<GridDim, BLOCK_HEIGHT>>>
      (x, a, y, stride, num_surfs, AxPy);

  hipDeviceSynchronize();

#ifdef GPU_PROF
  hipEventRecord(kernelStop, 0);
  hipEventSynchronize(kernelStop);
  hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
  gflopRate = (flops / 1e9) / (kernelTime / 1e3);
  fprintf(stderr, "AxPy kernel took %f ms @ %f Gflops\n", kernelTime, gflopRate);
#endif
}

__global__
void xvpbKernel(const float *x, const float *a, float y, int stride, int num_surfs, float *AxPy) {
  unsigned int blockOff, scalOff, off, scal;
  float xXReg, xYReg, xZReg, aReg, AxPyXReg, AxPyYReg, AxPyZReg;
  scalOff = blockIdx.x * stride;
  blockOff = scalOff * DIM;

  int numChunkLoop = stride / BLOCK_HEIGHT;

  off = blockOff + threadIdx.x;
  scal = scalOff + threadIdx.x;
  
  for (int chunk = 0; chunk < numChunkLoop; chunk++) {
    xXReg = x[off];
    xYReg = x[off + stride];
    xZReg = x[off + stride + stride];
    aReg = a[scal];
    

    AxPyXReg = aReg * xXReg + y;
    AxPyYReg = aReg * xYReg + y;
    AxPyZReg = aReg * xZReg + y;

    AxPy[off] = AxPyXReg;
    AxPy[off + stride] = AxPyYReg;
    AxPy[off + stride + stride] = AxPyZReg;

    off += BLOCK_HEIGHT;
    scal += BLOCK_HEIGHT;
  }

  if (off < blockOff + stride) {
    xXReg = x[off];
    xYReg = x[off + stride];
    xZReg = x[off + stride + stride];
    aReg = a[scal];
    

    AxPyXReg = aReg * xXReg + y;
    AxPyYReg = aReg * xYReg + y;
    AxPyZReg = aReg * xZReg + y;

    AxPy[off] = AxPyXReg;
    AxPy[off + stride] = AxPyYReg;
    AxPy[off + stride + stride] = AxPyZReg;
  }
}

void xvpbGpu(const float *x, const float *a, float y, int stride, int num_surfs, float *AxPy) {

#ifdef GPU_PROF
  float kernelTime, flops, gflopRate;
  flops = AXPY_FLOPS * stride * num_surfs;
  hipEvent_t kernelStart, kernelStop;
  hipEventCreate(&kernelStart);
  hipEventCreate(&kernelStop);
  hipEventRecord(kernelStart, 0);
#endif

  int GridDim = num_surfs;
  xvpbKernel<<<GridDim, BLOCK_HEIGHT>>>
      (x, a, y, stride, num_surfs, AxPy);

  hipDeviceSynchronize();

#ifdef GPU_PROF
  hipEventRecord(kernelStop, 0);
  hipEventSynchronize(kernelStop);
  hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
  gflopRate = (flops / 1e9) / (kernelTime / 1e3);
  fprintf(stderr, "AxPy kernel took %f ms @ %f Gflops\n", kernelTime, gflopRate);
#endif
}

__global__
void uyInvKernel(const float *x, const float *y, int stride, int num_surfs, float *xDy) {
  unsigned int blockOff, scalOff, off, scal;
  float xXReg, xYReg, xZReg, yReg, xDyXReg, xDyYReg, xDyZReg;
  scalOff = blockIdx.x * stride;
  blockOff = scalOff * DIM;

  int numChunkLoop = stride / BLOCK_HEIGHT;

  off = blockOff + threadIdx.x;
  scal = scalOff + threadIdx.x;
  
  for (int chunk = 0; chunk < numChunkLoop; chunk++) {
    xXReg = x[off];
    xYReg = x[off + stride];
    xZReg = x[off + stride + stride];
    yReg = y[scal];
    

    xDyXReg = xXReg / yReg;
    xDyYReg = xYReg / yReg;
    xDyZReg = xZReg / yReg;

    xDy[off] = xDyXReg;
    xDy[off + stride] = xDyYReg;
    xDy[off + stride + stride] = xDyZReg;

    off += BLOCK_HEIGHT;
    scal += BLOCK_HEIGHT;
  }

  if (off < blockOff + stride) {
    xXReg = x[off];
    xYReg = x[off + stride];
    xZReg = x[off + stride + stride];
    yReg = y[scal];
    

    xDyXReg = xXReg / yReg;
    xDyYReg = xYReg / yReg;
    xDyZReg = xZReg / yReg;

    xDy[off] = xDyXReg;
    xDy[off + stride] = xDyYReg;
    xDy[off + stride + stride] = xDyZReg;
  }
}

void uyInvGpu(const float *x, const float *y, int stride, int num_surfs, float *xDy) {

#ifdef GPU_PROF
  float kernelTime, flops, gflopRate;
  flops = XDY_FLOPS * stride * num_surfs;
  hipEvent_t kernelStart, kernelStop;
  hipEventCreate(&kernelStart);
  hipEventCreate(&kernelStop);
  hipEventRecord(kernelStart, 0);
#endif

  int GridDim = num_surfs;
  uyInvKernel<<<GridDim, BLOCK_HEIGHT>>>
      (x, y, stride, num_surfs, xDy);

  hipDeviceSynchronize();

#ifdef GPU_PROF
  hipEventRecord(kernelStop, 0);
  hipEventSynchronize(kernelStop);
  hipEventElapsedTime(&kernelTime, kernelStart, kernelStop);
  gflopRate = (flops / 1e9) / (kernelTime / 1e3);
  fprintf(stderr, "xDy kernel took %f ms @ %f Gflops\n", kernelTime, gflopRate);
#endif
}

__global__
void sqrtKernel(const float *x_in, int length, float *x_out) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    x_out[idx] = sqrtf(x_in[idx]);
  }
}

void SqrtGpu(const float* x_in, int stride, int num_surfs, float *x_out) {
  int length = stride * num_surfs;
  int grid = length / BLOCK_HEIGHT + 1;
  sqrtKernel<<<grid, BLOCK_HEIGHT>>> (x_in, length, x_out);
  hipDeviceSynchronize();
}

__global__
void invKernel(const float *x_in, int length, float *x_out) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    x_out[idx] = 1.0F / x_in[idx];
  }
}

void InvGpu(const float* x_in, int stride, int num_surfs, float *x_out) {
  int length = stride * num_surfs;
  int grid = length / BLOCK_HEIGHT + 1;
  invKernel<<<grid, BLOCK_HEIGHT>>> (x_in, length, x_out);
  hipDeviceSynchronize();
}

__global__
void xyKernel(const float *x_in, const float *y_in, int length, float *xy_out) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    xy_out[idx] = x_in[idx] * y_in[idx];
  }
}

void xyGpu(const float* x_in, const float *y_in, int stride, int num_surfs, float *xy_out) {
  int length = stride * num_surfs;
  int grid = length / BLOCK_HEIGHT + 1;
  xyKernel<<<grid, BLOCK_HEIGHT>>> (x_in, y_in, length, xy_out);
  hipDeviceSynchronize();
}

__global__
void xyInvKernel(const float *x_in, const float *y_in, int length, float *xDy_out) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    xDy_out[idx] = x_in[idx] / y_in[idx];
  }
}

void xyInvGpu(const float* x_in, const float *y_in, int stride, int num_surfs, float *xDy_out) {
  int length = stride * num_surfs;
  int grid = length / BLOCK_HEIGHT + 1;
  xyInvKernel<<<grid, BLOCK_HEIGHT>>> (x_in, y_in, length, xDy_out);
  hipDeviceSynchronize();
}

__global__
void reduceKernel(const float *x_in, const float *w_in, const float *q_in,
                   int stride, float *int_x_dw) {
  float threadSum = 0.0F;
  __shared__
  float sum[BLOCK_HEIGHT];

  int xOff = blockIdx.x * stride + threadIdx.x;
  int qOff = threadIdx.x;

  while(xOff < (blockIdx.x + 1)* stride) {
    threadSum += x_in[xOff] * w_in[xOff] * q_in[qOff];
    xOff += BLOCK_HEIGHT;
    qOff += BLOCK_HEIGHT;
  }
  sum[threadIdx.x] = threadSum;
  int redOff = 1;
  int redStride = 2;
  while(redOff != BLOCK_HEIGHT) {
    if (threadIdx.x % redStride == 0) {
      syncthreads();
      sum[threadIdx.x] += sum[threadIdx.x + redOff];
    }
    redOff = redStride;
    redStride *= 2;
  }
  if(threadIdx.x == 0) {
    int_x_dw[blockIdx.x] = sum[0];
  }
}


__global__
void reduceKernel(const float *w_in, const float *q_in,
                   int stride, float *int_x_dw) {
  float threadSum = 0.0F;
  __shared__
  float sum[BLOCK_HEIGHT];

  int xOff = blockIdx.x * stride + threadIdx.x;
  int qOff = threadIdx.x;

  while(xOff < (blockIdx.x + 1)* stride) {
    threadSum += w_in[xOff] * q_in[qOff];
    xOff += BLOCK_HEIGHT;
    qOff += BLOCK_HEIGHT;
  }
  sum[threadIdx.x] = threadSum;
  int redOff = 1;
  int redStride = 2;
  while(redOff != BLOCK_HEIGHT) {
    if (threadIdx.x % redStride == 0) {
      syncthreads();
      sum[threadIdx.x] += sum[threadIdx.x + redOff];
    }
    redOff = redStride;
    redStride *= 2;
  }
  if(threadIdx.x == 0) {
    int_x_dw[blockIdx.x] = sum[0];
  }
}


void ReduceGpu(const float *x_in, const float *w_in, const float *q_in,
                int stride, int num_surfs, float *int_x_dw) {
  int grid = num_surfs;
  if (x_in != NULL)
    reduceKernel<<<grid, BLOCK_HEIGHT>>> (x_in, w_in, q_in, stride, int_x_dw);
  else
    reduceKernel<<<grid, BLOCK_HEIGHT>>> (w_in, q_in, stride, int_x_dw);
  hipDeviceSynchronize();
}

void CircShiftGpu(const float *arr_in, int n_vecs, int vec_length, int shift, float *arr_out) {
  shift = shift % vec_length;
  if (shift < 0) {
    shift += vec_length;
  }
  int base_in, base_out;
  for (int ii = 0; ii < n_vecs; ii++) {
    base_out = ii * vec_length;
    base_in = base_out + vec_length - shift;
    hipMemcpy(arr_out + base_out, arr_in + base_in, sizeof(float) * shift, hipMemcpyDeviceToDevice);
    base_in = base_out;
    base_out += shift;
    hipMemcpy(arr_out + base_out, arr_in + base_in, sizeof(float) * (vec_length - shift), hipMemcpyDeviceToDevice);
  }
  hipDeviceSynchronize();
}

__global__
void axpyKernel(float a, const float *x, const float *y, int length, float *axpy) {

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    axpy[idx] = a * x[idx] + y[idx];
  }
}

void axpyGpu(float a, const float* x_in, const float *y_in, int stride, int num_surfs, float *axpy_out) {
  int length = stride * num_surfs;
  int grid = length / BLOCK_HEIGHT + 1;
  axpyKernel<<<grid, BLOCK_HEIGHT>>> (a, x_in, y_in, length, axpy_out);
}

__global__
void axpbKernel(float a, const float *x, const float b, int length, float *axpy) {

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    axpy[idx] = a * x[idx] + b;
  }
}

void axpbGpu(float a, const float* x_in, float b, int stride, int num_surfs, float *axpb_out) {
  int length = stride * num_surfs;
  int grid = length / BLOCK_HEIGHT + 1;
  axpbKernel<<<grid, BLOCK_HEIGHT>>> (a, x_in, b, length, axpb_out);
  hipDeviceSynchronize();
}

__global__
void shuffle(float *in, int m, int n, int dim, float *out) {
  int sub, add;
  int block_off = blockIdx.x * dim * m;

  in += block_off;
  out += block_off;

  int thread_off = threadIdx.x;
  int out_off;

  while(thread_off < dim * m) {
    int f = thread_off / m;
    sub = f * m;
    add = f;
    out_off = (thread_off - sub) * dim + add;
    out[out_off] = in[thread_off];

    thread_off += BLOCK_HEIGHT;
  }
}


void cuda_shuffle(float *in, int m, int n, int dim, float *out) {
  int grid = n;
  shuffle<<<grid, BLOCK_HEIGHT>>> (in, m, n, dim, out);
  hipDeviceSynchronize();
}

///
#define PI_8I 0.0397887358F

__global__
void stokes(int m, int n, int t_head, const float *T, const float *S, const float *D, float *U, const float *Q) {
  float3 trg_reg;
  float3 src_reg;
  float3 pot_reg;
  float3 dis_reg;
  float3 u_reg;
  __shared__
  float3 u_sh[BLOCK_HEIGHT];

  int t_off = blockIdx.x * 3 * m + t_head + blockIdx.y;

  trg_reg.x = T[t_off];
  trg_reg.y = T[m + t_off];
  trg_reg.z = T[m + m + t_off];

  u_reg = make_float3(0.0, 0.0, 0.0);

  int block_off = blockIdx.x * 3 * m + threadIdx.x;
  int s = threadIdx.x;

  while(block_off < blockIdx.x * 3 * m + m) {
    src_reg.x = S[block_off];
    src_reg.y = S[block_off + m];
    src_reg.z = S[block_off + m + m];

    pot_reg.x = D[block_off] * Q[s];
    pot_reg.y = D[block_off + m] * Q[s];
    pot_reg.z = D[block_off + m + m] * Q[s];

    dis_reg.x = src_reg.x - trg_reg.x;
    dis_reg.y = src_reg.y - trg_reg.y;
    dis_reg.z = src_reg.z - trg_reg.z;

    float inv_r = rsqrtf(dis_reg.x * dis_reg.x + dis_reg.y * dis_reg.y
                          + dis_reg.z * dis_reg.z);

    inv_r = inv_r + (inv_r-inv_r);
    inv_r = fmaxf(inv_r,0.0F);
    
    float tmp_scal = (dis_reg.x * pot_reg.x + dis_reg.y * pot_reg.y
                       + dis_reg.z * pot_reg.z) * inv_r * inv_r;
    pot_reg.x += tmp_scal * dis_reg.x;
    pot_reg.y += tmp_scal * dis_reg.y;
    pot_reg.z += tmp_scal * dis_reg.z;

    u_reg.x += pot_reg.x * inv_r;
    u_reg.y += pot_reg.y * inv_r;
    u_reg.z += pot_reg.z * inv_r;

    block_off += BLOCK_HEIGHT;
    s += BLOCK_HEIGHT;
  }

  u_sh[threadIdx.x].x = u_reg.x;
  u_sh[threadIdx.x].y = u_reg.y;
  u_sh[threadIdx.x].z = u_reg.z;

  int off = 1;
  int stride = 2;
  while (off != BLOCK_HEIGHT) {
    if (threadIdx.x % stride == 0) {
      syncthreads();
      u_sh[threadIdx.x].x += u_sh[threadIdx.x + off].x;
      u_sh[threadIdx.x].y += u_sh[threadIdx.x + off].y;
      u_sh[threadIdx.x].z += u_sh[threadIdx.x + off].z;
    }
    off = stride;
    stride *= 2;
  }
  if (threadIdx.x == 0) {
    U[t_off] = u_sh[0].x * PI_8I;
    U[m + t_off] = u_sh[0].y * PI_8I;
    U[m + m + t_off] = u_sh[0].z * PI_8I;
  }

}

__global__
void stokes(int m, int n, int t_head, const float *T, const float *S, const float *D, float *U) {
  float3 trg_reg;
  float3 src_reg;
  float3 pot_reg;
  float3 dis_reg;
  float3 u_reg;
  __shared__
  float3 u_sh[BLOCK_HEIGHT];

  int t_off = blockIdx.x * 3 * m + t_head + blockIdx.y;

  trg_reg.x = T[t_off];
  trg_reg.y = T[m + t_off];
  trg_reg.z = T[m + m + t_off];

  u_reg = make_float3(0.0, 0.0, 0.0);

  int block_off = blockIdx.x * 3 * m + threadIdx.x;
  int s = threadIdx.x;

  while(block_off < blockIdx.x * 3 * m + m) {
    src_reg.x = S[block_off];
    src_reg.y = S[block_off + m];
    src_reg.z = S[block_off + m + m];

    pot_reg.x = D[block_off];
    pot_reg.y = D[block_off + m];
    pot_reg.z = D[block_off + m + m];

    dis_reg.x = src_reg.x - trg_reg.x;
    dis_reg.y = src_reg.y - trg_reg.y;
    dis_reg.z = src_reg.z - trg_reg.z;

    float inv_r = rsqrtf(dis_reg.x * dis_reg.x + dis_reg.y * dis_reg.y
                          + dis_reg.z * dis_reg.z);

    inv_r = inv_r + (inv_r-inv_r);
    inv_r = fmaxf(inv_r,0.0F);
    
    float tmp_scal = (dis_reg.x * pot_reg.x + dis_reg.y * pot_reg.y
                       + dis_reg.z * pot_reg.z) * inv_r * inv_r;
    pot_reg.x += tmp_scal * dis_reg.x;
    pot_reg.y += tmp_scal * dis_reg.y;
    pot_reg.z += tmp_scal * dis_reg.z;

    u_reg.x += pot_reg.x * inv_r;
    u_reg.y += pot_reg.y * inv_r;
    u_reg.z += pot_reg.z * inv_r;

    block_off += BLOCK_HEIGHT;
    s += BLOCK_HEIGHT;
  }

  u_sh[threadIdx.x].x = u_reg.x;
  u_sh[threadIdx.x].y = u_reg.y;
  u_sh[threadIdx.x].z = u_reg.z;

  int off = 1;
  int stride = 2;
  while (off != BLOCK_HEIGHT) {
    if (threadIdx.x % stride == 0) {
      syncthreads();
      u_sh[threadIdx.x].x += u_sh[threadIdx.x + off].x;
      u_sh[threadIdx.x].y += u_sh[threadIdx.x + off].y;
      u_sh[threadIdx.x].z += u_sh[threadIdx.x + off].z;
    }
    off = stride;
    stride *= 2;
  }
  if (threadIdx.x == 0) {
    U[t_off] = u_sh[0].x * PI_8I;
    U[m + t_off] = u_sh[0].y * PI_8I;
    U[m + m + t_off] = u_sh[0].z * PI_8I;
  }

}


void cuda_stokes(int m, int n, int t_head, int t_tail, const float *T, const float *S, const float *D, float *U, const float *Q) {
  dim3 grid;
  grid.x = n;
  grid.y = t_tail - t_head;

  if (Q != NULL)
      stokes<<<grid, BLOCK_HEIGHT>>> (m, n, t_head, T, S, D, U, Q);
  else
      stokes<<<grid, BLOCK_HEIGHT>>> (m, n, t_head, T, S, D, U);
  hipDeviceSynchronize();
}

void ResampleGpu(int p, int n_funs, int q, const float *shc_p, float *shc_q) {

  float *out_deb = shc_q;
  int leg_order = p + 1;
  int new_leg_order = q + 1;
  int min_leg_order = (leg_order < new_leg_order) ? leg_order : new_leg_order;

  for(int v = 0; v < n_funs; v++) {
    hipMemcpy(shc_q, shc_p, sizeof(float) * min_leg_order, hipMemcpyDeviceToDevice);
    shc_q += min_leg_order;
    shc_p += min_leg_order;
    if (new_leg_order > leg_order) {
      hipMemset(shc_q, 0, sizeof(float) * (new_leg_order - leg_order));
      shc_q += (new_leg_order - leg_order);
    }
    if (leg_order > new_leg_order)
      shc_p += (leg_order - new_leg_order);
  }
  leg_order--;
  new_leg_order--;
  min_leg_order--;

  for(; min_leg_order > 1; min_leg_order--, leg_order--, new_leg_order--) {
    for(int v = 0; v < n_funs; v++) {
      hipMemcpy(shc_q, shc_p, sizeof(float) * min_leg_order, hipMemcpyDeviceToDevice);
      shc_q += min_leg_order;
      shc_p += min_leg_order;
      if (new_leg_order > leg_order) {
        hipMemset(shc_q, 0, sizeof(float) * (new_leg_order - leg_order));
        shc_q += (new_leg_order - leg_order);
      }
      if (leg_order > new_leg_order)
        shc_p += (leg_order - new_leg_order);
    }
    for(int v = 0; v < n_funs; v++) {
      hipMemcpy(shc_q, shc_p, sizeof(float) * min_leg_order, hipMemcpyDeviceToDevice);
      shc_q += min_leg_order;
      shc_p += min_leg_order;
      if (new_leg_order > leg_order) {
        hipMemset(shc_q, 0, sizeof(float) * (new_leg_order - leg_order));
        shc_q += (new_leg_order - leg_order);
      }
      if (leg_order > new_leg_order)
        shc_p += (leg_order - new_leg_order);
    }
  }

  for(int v = 0; v < n_funs; v++) {
    hipMemcpy(shc_q, shc_p, sizeof(float) * min_leg_order, hipMemcpyDeviceToDevice);
    shc_q += min_leg_order;
    shc_p += min_leg_order;
    if (new_leg_order > leg_order) {
      hipMemset(shc_q, 0, sizeof(float) * (new_leg_order - leg_order));
      shc_q += (new_leg_order - leg_order);
    }
    if (leg_order > new_leg_order)
      shc_p += (leg_order - new_leg_order);
  }

  leg_order--;
  new_leg_order--;
  min_leg_order--;

  float *outputs_end = out_deb + n_funs * q * (q + 2);
  if (shc_q < outputs_end) {
    hipMemset(shc_q, 0, sizeof(float) * (outputs_end - shc_q));
  }
  hipDeviceSynchronize();
}

__global__
void xyMKernel(const float *x_in, const float *y_in, int length, int stride, float *xy_out) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    xy_out[idx] = x_in[idx] * y_in[idx % stride];
  }
}

void xyMGpu(const float* x_in, const float *y_in, int stride, int num_surfs, float *xy_out) {
  int length = stride * num_surfs;
  int grid = length / BLOCK_HEIGHT + 1;
  xyMKernel<<<grid, BLOCK_HEIGHT>>> (x_in, y_in, length, stride, xy_out);
  hipDeviceSynchronize();
}

void ScaleFreqsGpu(int p, int n_funs, const float *shc_in, const float *alpha, float *shc_out) {
    int leg_order = p+1;
    xyMGpu(shc_in, alpha, leg_order, n_funs, shc_out);
    alpha += leg_order;
    shc_in += n_funs * leg_order;
    shc_out += n_funs * leg_order;
    leg_order--;

    // process remaining frequencies except the last cosine
    for (; leg_order>1; leg_order--) 
    {
        // first process cosine
        xyMGpu(shc_in, alpha, leg_order, n_funs, shc_out);
        alpha += leg_order;
        shc_in += n_funs * leg_order;
        shc_out += n_funs * leg_order;
        
        // then process sine
        xyMGpu(shc_in, alpha, leg_order, n_funs, shc_out);
        alpha += leg_order;
        shc_in += n_funs * leg_order;
        shc_out += n_funs * leg_order;
    }
    
    // process last cosine
    xyMGpu(shc_in, alpha, leg_order, n_funs, shc_out);
    hipDeviceSynchronize();
}

__global__
void avpwKernel(const float *a_in, const float *v_in, const float *w_in,
                 int size, int length, float *avpw_out) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < length) {
    avpw_out[idx] = a_in[idx / (size)] * v_in[idx] + w_in[idx];
  }
}


void avpwGpu(const float *a_in, const float *v_in, const float *w_in,
              int stride, int num_surfs, float *avpw_out) {
  dim3 grid;
  grid.x = num_surfs * stride * 3 / BLOCK_HEIGHT + 1;
  avpwKernel<<<grid, BLOCK_HEIGHT>>> (a_in, v_in, w_in, stride * 3, num_surfs * stride * 3, avpw_out);
  hipDeviceSynchronize();
}


__global__
void reduceMaxKernel(float *in, int n) {
  __shared__ float sdata[BLOCK_HEIGHT];
  int idx = blockIdx.x * BLOCK_HEIGHT + threadIdx.x;
  if (idx < n)
    sdata[threadIdx.x] = in[idx];
  else
    sdata[threadIdx.x] = -1e9;

  int redOff = 1;
  int redStride = 2;
  while(redOff != BLOCK_HEIGHT) {
    if (threadIdx.x % redStride == 0) {
      syncthreads();
      sdata[threadIdx.x] = fmaxf(sdata[threadIdx.x], sdata[threadIdx.x + redOff]);
    }
    redOff = redStride;
    redStride *= 2;
  }
  if(threadIdx.x == 0) {
    in[blockIdx.x] = sdata[0];
  }
}

float maxGpu(float *in, int n) {
  while(n > 0) {
    int grid = n / BLOCK_HEIGHT + 1;
    reduceMaxKernel<<<grid, BLOCK_HEIGHT>>> (in, n);
    n /= BLOCK_HEIGHT;
  }
  float max;
  hipMemcpy(&max, in, sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  return max;
}
